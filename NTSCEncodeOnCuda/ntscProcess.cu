#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);
extern int image_N;

float * dev_ref_luma;
float * dev_ref_chroma_u;
float * dev_ref_chroma_v;

__global__ void
ntsc_encode_frame(int N, float* luma, float* chroma_u, float* chroma_v, unsigned char* source) {
    // compute overall index from position of thread in current block,
    // and given the block we are in
    int refIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int line = blockIdx.x;
    int col = threadIdx.x;

    int imageOffset = (line * 514 + col) * 4;
    float im_R = source[imageOffset] / 256.0;
    float im_G = source[imageOffset+1] / 256.0;
    float im_B = source[imageOffset+2] / 256.0;
    float im_A = source[imageOffset+3] / 256.0;

    float im_Y = im_R * .299 + im_G * .587 + im_B * .114;
    float im_U = .492 * (im_B - im_Y);
    float im_V = .877 * (im_R - im_Y);
    //if (index > 1000 & index < 1100) printf("Index: %d\n, Block: %d, Dim: %d, Thread: %d",
    //        index, blockIdx.x, blockDim.x, threadIdx.x);

    int arrIndex = 635 * line + 106 + col;
    luma[arrIndex] = (im_Y * 0.7) + 0.3;
    chroma_u[arrIndex] = im_U;
    chroma_v[arrIndex] = im_V;
}

void
ntscCuda(int N, float* luma, float* chroma_u, float* chroma_v, unsigned char* source){

    int totalBytes = sizeof(float) * N;
    int totalBytesUChar = sizeof(unsigned char) * image_N;

    // compute number of blocks and threads per block
    //const int threadsPerBlock = 512;
    //const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int blocks = ((N / 635) - 20); //242 TV Lines
    const int threadsPerBlock = 514; //514 'pixels' per line

    float* device_luma;
    float* device_chroma_u;
    float* device_chroma_v;
    unsigned char* device_source;

    // start timing
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_luma, totalBytes);
    hipMalloc(&device_chroma_u, totalBytes);
    hipMalloc(&device_chroma_v, totalBytes);
    hipMalloc(&device_source, totalBytesUChar);

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_luma, dev_ref_luma, totalBytes, hipMemcpyDeviceToDevice);
    hipMemcpy(device_chroma_u, dev_ref_chroma_u, totalBytes, hipMemcpyDeviceToDevice);
    hipMemcpy(device_chroma_v, dev_ref_chroma_v, totalBytes, hipMemcpyDeviceToDevice);
    hipMemcpy(device_source, source, totalBytesUChar, hipMemcpyHostToDevice);

    // run kernel
    ntsc_encode_frame<<<blocks, threadsPerBlock>>>(N, device_luma, device_chroma_u, device_chroma_v, device_source);
    hipDeviceSynchronize();

    //
    // TODO copy result from GPU using hipMemcpy
    //
    hipMemcpy(luma, device_luma, totalBytes, hipMemcpyDeviceToHost);
    hipMemcpy(chroma_u, device_chroma_u, totalBytes, hipMemcpyDeviceToHost);
    hipMemcpy(chroma_v, device_chroma_v, totalBytes, hipMemcpyDeviceToHost);
    //hipMemcpy(source, source, totalBytes, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // TODO free memory buffers on the GPU
    hipFree(device_luma);
    hipFree(device_chroma_u);
    hipFree(device_chroma_v);
    hipFree(device_source);
}

void
load_reference_arrays(float * luma, float * chroma_u, float * chroma_v, int n){
    int totalBytes = n * sizeof(float);

    hipMalloc(&dev_ref_luma, totalBytes);
    hipMalloc(&dev_ref_chroma_u, totalBytes);
    hipMalloc(&dev_ref_chroma_v, totalBytes);

    hipMemcpy(dev_ref_luma, luma, totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_ref_chroma_u, chroma_u, totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_ref_chroma_v, chroma_v, totalBytes, hipMemcpyHostToDevice);
}

void
clear_reference_arrays(){
    hipFree(dev_ref_luma);
    hipFree(dev_ref_chroma_u);
    hipFree(dev_ref_chroma_v);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
