#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"

extern float toBW(int bytes, float sec);

__global__ void
ntsc_encode_frame(int N, float* luma, float* chroma_u, float* chroma_v, float* source) {

    // compute overall index from position of thread in current block,
    // and given the block we are in
    int refIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel = blockIdx.x;
    int line = threadIdx.x;
    //if (index > 1000 & index < 1100) printf("Index: %d\n, Block: %d, Dim: %d, Thread: %d",
    //        index, blockIdx.x, blockDim.x, threadIdx.x);

    int arrIndex = 635 * line + 106 + pixel;
    luma[arrIndex] = source[arrIndex];
    chroma_u[arrIndex] = (source[arrIndex] * 2 - 256) / 512.0f;
    chroma_v[arrIndex] = (source[arrIndex] * 1.5 - 256) / 512.0f;
}

void
ntscCuda(int N, float* luma, float* chroma_u, float* chroma_v, float* source) {

    int totalBytes = sizeof(float) * N;

    // compute number of blocks and threads per block
    //const int threadsPerBlock = 512;
    //const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    const int threadsPerBlock = ((N / 635) - 20); //242 TV Lines
    const int blocks = 514; //514 'pixels' per line

    float* device_luma;
    float* device_chroma_u;
    float* device_chroma_v;
    float* device_source;

    //
    // TODO allocate device memory buffers on the GPU using hipMalloc
    //
    hipMalloc(&device_luma, totalBytes);
    hipMalloc(&device_chroma_u, totalBytes);
    hipMalloc(&device_chroma_v, totalBytes);
    hipMalloc(&device_source, totalBytes);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    //
    // TODO copy input arrays to the GPU using hipMemcpy
    //
    hipMemcpy(device_luma, luma, totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_chroma_u, chroma_u, totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_chroma_v, chroma_v, totalBytes, hipMemcpyHostToDevice);
    hipMemcpy(device_source, source, totalBytes, hipMemcpyHostToDevice);

    // run kernel
    ntsc_encode_frame<<<blocks, threadsPerBlock>>>(N, device_luma, device_chroma_u, device_chroma_v, device_source);
    hipDeviceSynchronize();

    //
    // TODO copy result from GPU using hipMemcpy
    //
    hipMemcpy(luma, device_luma, totalBytes, hipMemcpyDeviceToHost);
    hipMemcpy(chroma_u, device_chroma_u, totalBytes, hipMemcpyDeviceToHost);
    hipMemcpy(chroma_v, device_chroma_v, totalBytes, hipMemcpyDeviceToHost);
    //hipMemcpy(source, source, totalBytes, hipMemcpyDeviceToHost);

    // end timing after result has been copied back into host memory
    double endTime = CycleTimer::currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    // TODO free memory buffers on the GPU
    hipFree(device_luma);
    hipFree(device_chroma_u);
    hipFree(device_chroma_v);
    hipFree(device_source);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
